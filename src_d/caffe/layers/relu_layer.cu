#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/relu_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ReLUForward(const int n, const Dtype* in, Dtype* out,
    Dtype negative_slope) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > 0 ? in[index] : in[index] * negative_slope;
  }
}

/////////////////////////////////////////////////////////////////////////// 1
template <typename Dtype>
__global__ void TransformData(const int n, const Dtype* in, char* relu_vec){
	CUDA_KERNEL_LOOP(index,n){relu_vec[index]=in[index]>0?1:0;}
}
/////////////////////////////////////////////////////////////////////////////

template <typename Dtype>
void ReLULayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  Dtype negative_slope = this->layer_param_.relu_param().negative_slope();
  // NOLINT_NEXT_LINE(whitespace/operators)
  ReLUForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data, negative_slope);
  CUDA_POST_KERNEL_CHECK;
  // << " count: " << count << " bottom_data: "
  //     << (unsigned long)bottom_data
  //     << " top_data: " << (unsigned long)top_data
  //     << " blocks: " << CAFFE_GET_BLOCKS(count)
  //     << " threads: " << CAFFE_CUDA_NUM_THREADS;

  /////////////////////////////////////////////////////////////////////// 2
    if(this->layer_param_.phase() == TRAIN){
  //	  LOG(INFO)<<"	forward: layer "<<this->layer_param_.name();
  	  if(!char_bottom_data_){
//  		  LOG(INFO)<<"alloc memory for char data";
  		  CUDA_CHECK(hipMalloc((void**)&char_bottom_data_, count*sizeof(char)));
  	  }
  	  TransformData<Dtype><<<CAFFE_GET_BLOCKS(count),CAFFE_CUDA_NUM_THREADS>>>(
  			  count, bottom_data, char_bottom_data_);
  	  CUDA_POST_KERNEL_CHECK;
    }
    ///////////////////////////////////////////////////////////////////////

}

template <typename Dtype>
__global__ void ReLUBackward(const int n, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff, Dtype negative_slope) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * ((in_data[index] > 0)
        + (in_data[index] <= 0) * negative_slope);
  }
}

//////////////////////////////////////////////////////////////////// 3
template <typename Dtype>
__global__ void ReLUBackward_char_data(const int n, const Dtype* in_diff,
    const char* in_data, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * (in_data[index] > 0);
  }
}
//////////////////////////////////////////////////////////////////////

template <typename Dtype>
void ReLULayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    Dtype negative_slope = this->layer_param_.relu_param().negative_slope();
    // NOLINT_NEXT_LINE(whitespace/operators)
    if(negative_slope){///////////////////////////////
    	ReLUBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
    			count, top_diff, bottom_data, bottom_diff, negative_slope);
    	CUDA_POST_KERNEL_CHECK;
    ///////////////////////////////////////////////////////////////////// 4
    } else{
	  ReLUBackward_char_data<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
		count, top_diff, char_bottom_data_, bottom_diff);
	  CUDA_POST_KERNEL_CHECK;
	}
    /////////////////////////////////////////////////////////////////////
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(ReLULayer);


}  // namespace caffe
